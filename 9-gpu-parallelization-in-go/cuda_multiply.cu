
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <iostream>

const int ELEMENTS = 1<<25;

__global__
void multiply(int j, float *a, float *b, float *c)
{
 for (int i = 0; i < j; i++)
      c[i] = a[i] * b[i];
}

int main(void)
{

float *a, *b, *c;

hipMallocManaged(&a, ELEMENTS*sizeof(float));
hipMallocManaged(&b, ELEMENTS*sizeof(float));
hipMallocManaged(&c, ELEMENTS*sizeof(float));

for(int i = 0; i < ELEMENTS; i++)
{
    a[i] = 1.0; //rand() % 10;
    b[i] = 2.0; //rand() % 10;

}

multiply<<<1, 1>>>(ELEMENTS, a, b, c);


// Wait for GPU to finish before accessing on host
hipDeviceSynchronize();

//for (int k = 0; k < ELEMENTS; k++)
//{
//    std::cout << k << ":" << a[k] << "*" << b[k] << "=" << c[k] << "\n";
//}

hipFree(a);
hipFree(b);
hipFree(c);

return 0;

}
